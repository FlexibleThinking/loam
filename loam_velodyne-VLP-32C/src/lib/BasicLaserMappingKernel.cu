#include "hip/hip_runtime.h"
#include "loam_velodyne/BasicLaserMappingKernel.cuh"
#include <stdio.h>
CudaTest::CudaTest(void)
{
}
CudaTest::~CudaTest(void)
{
}

__global__ void sum_kernel(int a, int b, int *c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a + b;
}
int CudaTest::sum_cuda(int a, int b, int *c)
{
	int *f;
	hipMalloc((void**)&f, sizeof(int)* 1);
	hipMemcpy(f, c, sizeof(int)* 1, hipMemcpyHostToDevice);

	sum_kernel <<<1, 1>>>(a, b, f);

	hipMemcpy(c, f, sizeof(int) *1, hipMemcpyDeviceToHost);

	hipFree(f);

	return true;
}
